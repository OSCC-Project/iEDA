#include "hip/hip_runtime.h"
// ***************************************************************************************
// Copyright (c) 2023-2025 Peng Cheng Laboratory
// Copyright (c) 2023-2025 Institute of Computing Technology, Chinese Academy of
// Sciences Copyright (c) 2023-2025 Beijing Institute of Open Source Chip
//
// iEDA is licensed under Mulan PSL v2.
// You can use this software according to the terms and conditions of the Mulan
// PSL v2. You may obtain a copy of Mulan PSL v2 at:
// http://license.coscl.org.cn/MulanPSL2
//
// THIS SOFTWARE IS PROVIDED ON AN "AS IS" BASIS, WITHOUT WARRANTIES OF ANY
// KIND, EITHER EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO
// NON-INFRINGEMENT, MERCHANTABILITY OR FIT FOR A PARTICULAR PURPOSE.
//
// See the Mulan PSL v2 for more details.
// ***************************************************************************************
/**
 * @file ir_solver.cu
 * @author simin tao (taosm@pcl.ac.cn)
 * @brief The ir cuda solver.
 * @version 0.1
 * @date 2025-04-19
 *
 */

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <hipsolver.h>
#include <hipsparse.h>

#include <Eigen/Dense>
#include <Eigen/Sparse>
#include <Eigen/SparseLU>
#include <cmath>
#include <iostream>
#include <vector>

#include "gpu/cuda_common.cuh"
#include "gpu/kernel_common.h"
#include "ir_solver.cuh"

namespace iir {

/**
 * @brief The ir cg solver use cuda.
 * 
 * @param A 
 * @param b 
 * @param x0 
 * @param tol 
 * @param max_iter 
 * @return std::vector<double> 
 */
std::vector<double> ir_cg_solver(Eigen::SparseMatrix<double> &A,
                                 Eigen::VectorXd &b, Eigen::VectorXd &x0,
                                 const double tol, const int max_iter) {
  // Convert Eigen sparse matrix to CSR format
  A.makeCompressed();
  int num_rows = A.rows();
  int num_cols = A.cols();
  int nnz = A.nonZeros();
  const int *csrRowPtr = A.outerIndexPtr();
  const int *csrColInd = A.innerIndexPtr();
  const double *csrVal = A.valuePtr();

  // Allocate device memory
  double *d_csrVal, *d_b, *d_x, *d_r, *d_p, *d_Ap;
  int *d_csrRowPtr, *d_csrColInd;
  CUDA_CHECK(hipMalloc((void **)&d_csrVal, nnz * sizeof(double)));
  CUDA_CHECK(hipMalloc((void **)&d_csrRowPtr, (num_rows + 1) * sizeof(int)));
  CUDA_CHECK(hipMalloc((void **)&d_csrColInd, nnz * sizeof(int)));
  CUDA_CHECK(hipMalloc((void **)&d_b, num_rows * sizeof(double)));
  CUDA_CHECK(hipMalloc((void **)&d_x, num_rows * sizeof(double)));
  CUDA_CHECK(hipMalloc((void **)&d_r, num_rows * sizeof(double)));
  CUDA_CHECK(hipMalloc((void **)&d_p, num_rows * sizeof(double)));
  CUDA_CHECK(hipMalloc((void **)&d_Ap, num_rows * sizeof(double)));

  // Copy data to device
  CUDA_CHECK(hipMemcpy(d_csrVal, csrVal, nnz * sizeof(double), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_csrRowPtr, csrRowPtr, (num_rows + 1) * sizeof(int),
             hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_csrColInd, csrColInd, nnz * sizeof(int), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_b, b.data(), num_rows * sizeof(double), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_x, x0.data(), num_rows * sizeof(double), hipMemcpyHostToDevice));
  
  // for debug
  // print_device_array(d_b, num_rows);
  // print_device_array(d_x, num_rows);

  // cuSPARSE handle
  hipsparseHandle_t handle;
  hipsparseCreate(&handle);

  // Create matrix descriptor
  hipsparseSpMatDescr_t matA;
  hipsparseCreateCsr(&matA, num_rows, num_cols, nnz, d_csrRowPtr, d_csrColInd,
                    d_csrVal, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                    HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);

  // Create vector descriptors
  hipsparseDnVecDescr_t vecX, vecR, vecP, vecAp;
  hipsparseCreateDnVec(&vecX, num_cols, d_x, HIP_R_64F);
  hipsparseCreateDnVec(&vecR, num_rows, d_r, HIP_R_64F);
  hipsparseCreateDnVec(&vecP, num_rows, d_p, HIP_R_64F);
  hipsparseCreateDnVec(&vecAp, num_rows, d_Ap, HIP_R_64F);

  // Temporary variables
  double alpha = 1.0, beta = 0.0, r_dot_r, r_dot_r_new;

  // Initialize r = A * x
  size_t bufferSize = 0;
  hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha,
                          matA, vecX, &beta, vecR, HIP_R_64F,
                          HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize);
  void *dBuffer;
  hipMalloc(&dBuffer, bufferSize);

  hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecX,
               &beta, vecR, HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer);
  
  // for debug
  // print_csr_matrix(num_rows, num_cols, nnz, d_csrRowPtr, d_csrColInd, d_csrVal);
  // print_device_array(d_r, num_rows);

  hipblasHandle_t cublasHandle;
  hipblasCreate(&cublasHandle);

  // Perform r = b - A * x
  double neg_one = -1.0;
  hipblasDaxpy(cublasHandle, num_rows, &neg_one, d_r, 1, d_b, 1);  // r = b - vecR
  CUDA_CHECK(hipMemcpy(d_r, d_b, num_rows * sizeof(double), hipMemcpyDeviceToDevice)); // Copy b to r

  // for debug
  // print_device_array(d_r, num_rows);

  // Copy r to p
  CUDA_CHECK(hipMemcpy(d_p, d_r, num_rows * sizeof(double), hipMemcpyDeviceToDevice));

  // Compute initial r_dot_r
  hipblasDdot(cublasHandle, num_rows, d_r, 1, d_r, 1, &r_dot_r);

  int k = 0;
  double one = 1.0;
  double zero = 0.0;
  while (k < max_iter && sqrt(r_dot_r) > tol) {
    // Ap = A * p
    hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, matA, vecP,
                 &zero, vecAp, HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer);

    // alpha = r_dot_r / (p^T * Ap)
    double p_dot_Ap;
    hipblasDdot(cublasHandle, num_rows, d_p, 1, d_Ap, 1, &p_dot_Ap);
    alpha = r_dot_r / p_dot_Ap;

    // for debug
    // print_device_array(d_x, num_rows);
    // print_device_array(d_p, num_rows);

    // x = x + alpha * p
    hipblasDaxpy(cublasHandle, num_rows, &alpha, d_p, 1, d_x, 1);
    
    // for debug
    // print_device_array(d_x, num_rows);

    // r = r - alpha * Ap
    double neg_alpha = -alpha;
    hipblasDaxpy(cublasHandle, num_rows, &neg_alpha, d_Ap, 1, d_r, 1);

    // r_dot_r_new = r^T * r
    hipblasDdot(cublasHandle, num_rows, d_r, 1, d_r, 1, &r_dot_r_new);

    // beta = r_dot_r_new / r_dot_r
    beta = r_dot_r_new / r_dot_r;

    // p = r + beta * p
    hipblasDscal(cublasHandle, num_rows, &beta, d_p, 1);
    double one = 1.0;
    hipblasDaxpy(cublasHandle, num_rows, &one, d_r, 1, d_p, 1);

    r_dot_r = r_dot_r_new;
    k++;
  }
  
  CUDA_LOG_INFO("CG Iterations: %d", k - 1);
  CUDA_LOG_INFO("Final Residual Norm: %f", sqrt(r_dot_r));

  // for debug
  // print_device_array(d_x, num_rows);

  // Copy result back to host
  std::vector<double> x(num_rows);
  CUDA_CHECK(hipMemcpy(x.data(), d_x, num_rows * sizeof(double), hipMemcpyDeviceToHost));

  // Free resources
  hipFree(d_csrVal);
  hipFree(d_csrRowPtr);
  hipFree(d_csrColInd);
  hipFree(d_b);
  hipFree(d_x);
  hipFree(d_r);
  hipFree(d_p);
  hipFree(d_Ap);
  hipFree(dBuffer);
  hipsparseDestroySpMat(matA);
  hipsparseDestroyDnVec(vecX);
  hipsparseDestroyDnVec(vecR);
  hipsparseDestroyDnVec(vecAp);
  hipsparseDestroy(handle);
  hipblasDestroy(cublasHandle);

  CUDA_CHECK_ERROR();

  return x;
}

}  // namespace iir